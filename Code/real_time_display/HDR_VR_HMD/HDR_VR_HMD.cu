#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include <iostream>
using namespace std;
#define SIGMA 20	//was 20, 10 works fine
#define DENOM_C 49.f/(2.f*3.1415926f*SIGMA*SIGMA)*6.5		//consider 7x7 rectangular area of the LED light diode


// all on device memory
float4 *dM = NULL;	//Modulation layer, size is m-by-n
float4 *dP = NULL;	//Primary layer, size is (m/K)-by-(n/K) where K is the backlight ratio
float4 *dRec = NULL;	//Temporary reconstruction contributions, dimension is  m-by-n-by-3
float4 *dBL = NULL;	//Temporary storage for diffused backlight layer (immediately behind LCD)
float4 *dImageRGB  = NULL, *dImageXYZ = NULL;   //original image
float4 *dTemp = NULL;
size_t pitch;
texture<float4, 2, hipReadModeElementType> Target;


__global__ void ShowModulation(float4 *od, float4* M, int w, int h);
__global__ void ShowBacklight(float4 *od, float4* BL, int w, int h);
__global__ void BuildRecFromSimulatedBL(float4 *Rec, float4* M, float4* BL, int w, int h);
__global__ void InitializeTargetImage(float4* Target, int ImageWidth, int ImageHeight, int K); // Image edges darkening to avoid overfitting of the primary...
__global__ void SolveForModulationWithBL(float4* Rec, float4* M, float4* BL, int w, int h, int K);
__global__ void SolveForPrimary(float4* Rec, int ImageWidth, int ImageHeight, int K, float4* M, float4* P);
__global__ void UpdatePrimary(float4* P, float4* Update, int ImageWidth, int ImageHeight, int K);	//ONLY USED in WIDE_GUARDBAND

//TODO: This function may not be necessary
__device__ inline float RGB2Gray(float4 rgb);

__device__ inline float4 XYZ2RGB(float4 xyz);
__device__ inline float4 RGB2XYZ(float4 rgb);
__global__ void ShowRGBfromXYZ(float4* target, float4* source, int w, int h);
__global__ void ShowXYZfromRGB(float4* target, float4* source, int w, int h);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Suppose to get dM = rgb2gray(target), dM2 = 1-dM, dP = chroma(low_res_target), dP2 = 1 - dP
__global__ void InitializeM(float4 *M, int w, int h, int K);
__global__ void InitializeP(float4 *P, int w, int h, int K);	//w and h are in Primary array unit
__global__ void SimulateBacklight(float4* Backlight, float4* P, int ImageWidth, int ImageHeight, int K, int LEDRadius);

__device__ inline float Gaussian(int2 Dist);
__device__ float4 saturate(float4 val);// Overload with float4


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Nonnegative Matrix Factorization code below
__global__ void SolveForModulation(float4* Rec, float4* Ixyz, float4* M, float4* BL, int w, int h, int K, float Lambda)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= w || y >= h)	return;
	// Calculate residual, which is rec+Rec2-Target. Also retrieve the diffused backlight corresponding to the pixel
	float4 rec			= Rec[y*w + x];
	float4 residual		= rec - Ixyz[y*w + x];
	float4 backlight	= BL[y*w + x];

	// TODO: Figure out the vectorization here
	// M -= (PtR)/(PtP)
	float4 m		= M[y*w + x];
	float4 nom	= backlight*residual;
	float4 denom = (backlight*backlight) + 1e-3;
	m -= nom / denom;

	// Finally saturate the value so that it's physically plausible. Note need to do qualtization for low-bits
	M[y*w + x] = saturate(m);
}

__global__ void SolveForPrimary(float4* Rec, float4* Ixyz, int ImageWidth, int ImageHeight, int K, float4* M, float4* P)
{
	//32*32 = 1024 is the maximum number of thread supported, xyz for nom and w for denom!
	__shared__ float4 cache[32 * 32];			const float4 zero ={ 0 };
	cache[threadIdx.y * 32 + threadIdx.x] = zero;
	__syncthreads();

	int2 LedUV		= make_int2(int(blockIdx.x), int(blockIdx.y));
	int2 LocalXY	= (make_int2(threadIdx.x, threadIdx.y) - 16)*3.0;	//sample more sparsely for realtime perf.

	// The global coordinate of the pixel thread. The thread can be outside the image boundary, be careful.
	int2 GlobalXY	=  LocalXY + (LedUV*K + K / 2);	//local coordinate plus LED center coordinate	

	if (GlobalXY.x >= 0 && GlobalXY.x < ImageWidth && GlobalXY.y >= 0 && GlobalXY.y < ImageHeight)
	{
		// Start Processing
		float4	rec		= Rec[GlobalXY.y*ImageWidth + GlobalXY.x];
		float4	residual= rec - Ixyz[GlobalXY.y*ImageWidth + GlobalXY.x];

		float4	m		= M[GlobalXY.y*ImageWidth + GlobalXY.x];
		float	weight	= Gaussian(LocalXY);

		//somehow the XYZ2RGB isnt very effective to bring the residual from XYZ2RGB...
		cache[threadIdx.y * 32 + threadIdx.x]		= XYZ2RGB(residual*m*weight);	//Nominator
		cache[threadIdx.y * 32 + threadIdx.x].w	= (m.x*m.x + m.y*m.y + m.z*m.z + m.w*m.w)*weight + 1e-3;	//Nominator
	}

	// Start 2-dimensional reduction, make sure every step from now on is synchronized
	__syncthreads();
	for (int stride = 16; stride >= 1; stride/=2)
	{
		if (threadIdx.x < stride && threadIdx.y < stride)
		{
			cache[threadIdx.y * 32 + threadIdx.x] += cache[(threadIdx.y) * 32 + (threadIdx.x + stride)];	//2X, 1Y
			cache[threadIdx.y * 32 + threadIdx.x] += cache[(threadIdx.y + stride) * 32 + (threadIdx.x)];	//1X, 2Y
			cache[threadIdx.y * 32 + threadIdx.x] += cache[(threadIdx.y + stride) * 32 + (threadIdx.x + stride)];	//2X, 2Y
		}
		__syncthreads();
	}

	// Now only thread 0 represent the LED diode; However, if more blocks are used, we need atomicadd cache[0]
	// And also use the LED_ID to uniquely identify the only thread.
	if (threadIdx.x == 0 && threadIdx.y == 0)	//Only the first thread process the remaining data
	{
		//now update Primary
		float4 p = P[LedUV.y*(ImageWidth / K) + LedUV.x];
		p -=  cache[0]/ (cache[0].w + 1e-6);
		P[LedUV.y*(ImageWidth / K) + LedUV.x] = saturate(p);
	}
}


int		FirstTime = 1;
bool	InitializedX = false;

extern "C"
void InitializeSolverMemory(int width, int height, int ratio)
{
	hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
	checkCudaErrors(hipBindTexture2D(0, Target, dImageRGB, desc, width, height, pitch));	//bind dImageRGB to rgbaTex
	dim3 GridSizeP(((width / ratio) + 32 - 1) / 32, ((height / ratio) + 32 - 1) / 32);
	dim3 GridSizeM((width + 32 - 1) / 32, (height + 32 - 1) / 32);
	dim3 BlockSize(32, 32);

	InitializeP <<< GridSizeP, BlockSize >>>(dP, width / ratio, height / ratio, ratio);
	InitializeM <<< GridSizeM, BlockSize >>>(dM, width, height, ratio);

	SimulateBacklight <<< GridSizeM, BlockSize >>>	(dBL, dP, width, height, ratio, 6);
	BuildRecFromSimulatedBL <<< GridSizeM, BlockSize >>>	(dRec, dM, dBL, width, height);		//just initialization

	InitializedX = false;
}


extern "C"
double HdrVrHmdSolver(int* LAB_PHASE, int* STEP, float4 *dDest, int width, int height, int ratio, StopWatchInterface *timer, int val, float lambda)
{
	hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();

	dim3 GridSizeP(width / ratio, height / ratio);	//the grid size should be the number of the LEDs used in the solver ..... times guard-band size

	dim3 GridSizeM((width + 32 - 1) / 32, (height + 32 - 1) / 32);
	dim3 BlockSize(32, 32);

	if (FirstTime)
	{
		InitializeSolverMemory(width, height, ratio);
		FirstTime = 0;
	}

	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Solve the NMF problem for Modulation and Primary	
	{
		// solve for Modulation, 2 inner iteration acceleration global convergence
		for (int InnerIter = 0; InnerIter < 2; InnerIter++)
		{
			SolveForModulation <<< GridSizeM, BlockSize >>>	(dRec, dImageXYZ, dM, dBL, width, height, ratio, lambda);	//rank 1
			BuildRecFromSimulatedBL <<< GridSizeM, BlockSize >>>	(dRec, dM, dBL, width, height);		//Rebuild Reconstruction 1
		}

		// solve for Primaries, 2 inner iteration acceleration global convergence
		for (int InnerIter = 0; InnerIter < 2; InnerIter++)
		{
			SolveForPrimary <<< GridSizeP, BlockSize >>>	(dRec, dImageXYZ, width, height, ratio, dM, dP);
			SimulateBacklight <<< GridSizeM, BlockSize >>>	(dBL, dP, width, height, ratio, 6);
			BuildRecFromSimulatedBL <<< GridSizeM, BlockSize >>>	(dRec, dM, dBL, width, height);		//Rebuild Reconstruction 1
		}
	}

	checkCudaErrors(hipUnbindTexture(Target));

	// Switch what to show on screen.
	switch (val)
	{
	case 0:		ShowRGBfromXYZ <<<GridSizeM, BlockSize >>>		(dDest, dRec, width, height);								break;
	case 1:		ShowRGBfromXYZ <<<GridSizeM, BlockSize >>>		(dDest, dImageXYZ, width, height);								break;
	case 2:		checkCudaErrors(hipMemcpy(dDest, dM, sizeof(float4)*width*height, hipMemcpyDeviceToDevice)); break;
	case 3:		checkCudaErrors(hipMemcpy(dDest, dBL, sizeof(float4)*width*height, hipMemcpyDeviceToDevice)); break;
	}

	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Helper functions
__device__ inline float Gaussian(int2 Dist)
{
	Dist *= Dist;
	return DENOM_C*exp(-(Dist.x + Dist.y) / (2.f*SIGMA*SIGMA));
}

// Overload with float4
__device__ float4 saturate(float4 val)
{
	return make_float4(saturate(val.x), saturate(val.y), saturate(val.z), saturate(val.w));
}

__global__ void ShowModulation(float4 *od, float4* M, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	od[y*w + x] = M[y*w + x];
}

__global__ void ShowBacklight(float4* od, float4* BL, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	od[y*w + x] = BL[y*w + x];
	
}

__global__ void BuildRecFromSimulatedBL(float4 *Rec, float4* M, float4* BL, int w, int h)
{
	int2 GlobalXY = make_int2(blockIdx.x*blockDim.x + threadIdx.x, blockIdx.y*blockDim.y + threadIdx.y);
	// Note since we solve for the primary in RGB space, but the objective function is in XYZ space.
	Rec[GlobalXY.y*w + GlobalXY.x] = RGB2XYZ(M[GlobalXY.y*w + GlobalXY.x] * BL[GlobalXY.y*w + GlobalXY.x]);
}

// Image edges darkening to avoid overfitting of the primary...
__global__ void InitializeTargetImage(float4* Target, int ImageWidth, int ImageHeight, int K)
{
	int2 GlobalXY	= make_int2(blockIdx.x*blockDim.x + threadIdx.x, blockIdx.y*blockDim.y + threadIdx.y);

	if (GlobalXY.x < K || GlobalXY.x >= ImageWidth - K || GlobalXY.y < K || GlobalXY.y >= ImageHeight - K)
	{
		float4 Pixel = Target[GlobalXY.y * ImageWidth + GlobalXY.x];
		float dX = (K - GlobalXY.x  > GlobalXY.x - (ImageWidth - K)) ? K - GlobalXY.x : GlobalXY.x - (ImageWidth - K);
		float dY = (K - GlobalXY.y  > GlobalXY.y - (ImageHeight - K)) ? K - GlobalXY.y : GlobalXY.y - (ImageHeight - K);
		float d = (dX > dY) ? dX : dY;
		float	spread	= exp(-(d*d) / (2.f*SIGMA*SIGMA));
		Target[GlobalXY.y * ImageWidth + GlobalXY.x] = Pixel*spread;
	}
}

extern "C"
void freeTextures()
{
	//Data array
	checkCudaErrors(hipFree(dM));
	checkCudaErrors(hipFree(dP));

	//Intermediate variables
	checkCudaErrors(hipFree(dRec));
	checkCudaErrors(hipFree(dBL));
	checkCudaErrors(hipFree(dImageRGB));
	checkCudaErrors(hipFree(dImageXYZ));

	checkCudaErrors(hipFree(dTemp));
}

extern "C"
void initTexture(int width, int height, int K, float *hImage, float *hSubImage)
{
	// Allocatte data array
	checkCudaErrors(hipMalloc(&dM, sizeof(float4)*width*height));
	checkCudaErrors(hipMalloc(&dP, sizeof(float4)*(width)*(height)));	//allocate smaller memory seems to cause problem, perhaps MallocPitch?

	// Allocatte intermediate variables
	checkCudaErrors(hipMalloc(&dRec, sizeof(float4)*width*height));
	checkCudaErrors(hipMalloc(&dBL, sizeof(float4)*width*height));

	// Create target image and apply edge darkening to avoid over fitting problem
	checkCudaErrors(hipMallocPitch(&dImageRGB, &pitch, sizeof(float4)*width, height));
	checkCudaErrors(hipMallocPitch(&dImageXYZ, &pitch, sizeof(float4)*width, height));

	checkCudaErrors(hipMemcpy2D(dImageRGB, pitch, hImage, sizeof(float4)*width, sizeof(float4)*width, height, hipMemcpyHostToDevice));
	dim3 GridSizeM((width + 32 - 1) / 32, (height + 32 - 1) / 32);
	dim3 BlockSize(32, 32);
	InitializeTargetImage <<< GridSizeM, BlockSize >>>	(dImageRGB, width, height, K);
	ShowXYZfromRGB <<< GridSizeM, BlockSize >>>	(dImageXYZ, dImageRGB, width, height);

	checkCudaErrors(hipMalloc(&dTemp, sizeof(float4)*width*height));
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Suppose to get dM = rgb2gray(target), dM2 = 1-dM, dP = chroma(low_res_target), dP2 = 1 - dP
__global__ void InitializeM(float4* M, int w, int h, int K)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= w || y >= h)	return;
	M[y*w + x] = tex2D(Target, x + 0.5, y + 0.5);
}

__global__ void InitializeP(float4 *P, int w, int h, int K)	//w and h are in Primary array unit
{
	int u = blockIdx.x*blockDim.x + threadIdx.x;
	int v = blockIdx.y*blockDim.y + threadIdx.y;
	if (u >= w || v >= h)		return;

	float4 one = {1, 1, 1, 1};
	float4 zero = {0};
	//float4 pixel = tex2D(Target, u*K + K/2+0.5, v*K + K/2+0.5);
	float4 pixel = zero;
	for (int r = -K / 2; r <= K / 2; r++)
	{
		for (int c = -K / 2; c <= K / 2; c++)
		{
						pixel += tex2D(Target, u*K + K/2+0.5 + r, v*K + K/2+0.5 + c) / (K*K);
		}
	}

	P[v*w + u] = (pixel + 1.0)*0.5;
}

__global__ void SimulateBacklight(float4* Backlight, float4* P, int ImageWidth, int ImageHeight, int K, int LEDRadius)
{
	//32*32 = 1024 is the maximum number of thread supported, xyz for nom and w for denom!
	__shared__ float3 cache[64 * 64];	//(ImageWidth/K)*(ImageHeight/K)			
	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].x = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].x;	//x,y
	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].y = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].y;	//x,y
	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].z = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+0].z;	//x,y

	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].x = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].x;	//x+1,y
	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].y = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].y;	//x+1,y
	cache[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].z = P[(threadIdx.y*2+0)*64 + threadIdx.x*2+1].z;	//x+1,y

	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].x = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].x;	//x,y+1
	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].y = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].y;	//x,y+1
	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].z = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+0].z;	//x,y+1

	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].x = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].x;	//x+1,y+1
	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].y = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].y;	//x+1,y+1
	cache[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].z = P[(threadIdx.y*2+1)*64 + threadIdx.x*2+1].z;	//x+1,y+1
	// cache[(threadIdx.y * 2 + 0) * 64 + threadIdx.x * 2 + 0] = P[(threadIdx.y * 2 + 0) * 64 + threadIdx.x * 2 + 0];	//x,y
	// cache[(threadIdx.y * 2 + 0) * 64 + threadIdx.x * 2 + 1] = P[(threadIdx.y * 2 + 0) * 64 + threadIdx.x * 2 + 1];	//x+1,y
	// cache[(threadIdx.y * 2 + 1) * 64 + threadIdx.x * 2 + 0] = P[(threadIdx.y * 2 + 1) * 64 + threadIdx.x * 2 + 0];	//x,y+1
	// cache[(threadIdx.y * 2 + 1) * 64 + threadIdx.x * 2 + 1] = P[(threadIdx.y * 2 + 1) * 64 + threadIdx.x * 2 + 1];	//x+1,y+1


	__syncthreads();

	//USE SHARED MEMORY TO ACCELERATE THIS PART, SINCE 7x7 = 49 access is simply too much...
	int2 GlobalXY	= make_int2(blockIdx.x*blockDim.x + threadIdx.x, blockIdx.y*blockDim.y + threadIdx.y);
	if (GlobalXY.x >= ImageWidth || GlobalXY.y >= ImageHeight)	return;

	int2 NeighborUV;

	// Diffusion based backlight
	float4 AccumulatedPixel = {0};
	for (int r = -3; r <= 3; r++)
	{
		for (int c = -3; c <= 3; c++)
		{
			// Locate the neighbor LED indices and their center locations
			NeighborUV.x = int(GlobalXY.x / K) + c; 	NeighborUV.y = int(GlobalXY.y / K) + r;

			// Calculate the corresponding gaussian weighting/spreading. Outside boundaries doesn't count
			float spread = Gaussian(NeighborUV*K + K / 2 - GlobalXY);
			if (NeighborUV.x >= 0 && NeighborUV.x < ImageWidth / K && NeighborUV.y >= 0 && NeighborUV.y < ImageHeight / K)
			{
				float3 cacheval = cache[NeighborUV.y*(ImageWidth / K) + NeighborUV.x];

				AccumulatedPixel.x += spread*cacheval.x;	
				AccumulatedPixel.y += spread*cacheval.y;	
				AccumulatedPixel.z += spread*cacheval.z;
			}
		}
	}
	Backlight[GlobalXY.y*ImageWidth + GlobalXY.x] = (AccumulatedPixel);
}


__device__ inline float RGB2Gray(float4 rgb)
{
	return (0.2989 * rgb.x + 0.5870 * rgb.y + 0.1140 * rgb.z)*0.8 + 0.2;	//rgb2gray in Matlab
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// COLOR TRANSFORM FUNCTIONS

//D65	
//Matlab, note CIEXYZ can be > 1, so you need to divide by white point
// 
__device__ inline float4 XYZ2RGB(float4 xyz)
{
	float4 rgb ={ 0 };
	const float4 Row1 ={ 1, 0, 0, 0 };	//Optimization in RGB space instead of XYZ
	const float4 Row2 ={ 0, 1, 0, 0 };
	const float4 Row3 ={ 0, 0, 1, 0 };

	//sRGB D65
	//const float4 Row1 = { 3.2406,-1.5372,-0.4986, 0};
	//const float4 Row2 = {-0.9689, 1.8758, 0.0415, 0};
	//const float4 Row3 = { 0.0557,-0.2040, 1.0570, 0};

	rgb.x = dot(Row1, xyz);
	rgb.y = dot(Row2, xyz);
	rgb.z = dot(Row3, xyz);
	return rgb;
}

__device__ inline float4 RGB2XYZ(float4 rgb)
{
	float4 xyz ={ 0 };

	const float4 Row1 ={ 1, 0, 0, 0 };	//Optimization in RGB space instead of XYZ
	const float4 Row2 ={ 0, 1, 0, 0 };
	const float4 Row3 ={ 0, 0, 1, 0 };

	//sRGB D65
	//const float4 Row1 = {0.4124, 0.3576, 0.1805, 0};
	//const float4 Row2 = {0.2126, 0.7152, 0.0722, 0};
	//const float4 Row3 = {0.0193, 0.1192, 0.9505, 0};

	xyz.x = dot(Row1, rgb);
	xyz.y = dot(Row2, rgb);
	xyz.z = dot(Row3, rgb);
	return xyz;
}

// RGB2XYZ. Assuming WhitePoint D65
// http://www.brucelindbloom.com/index.html?Eqn_RGB_XYZ_Matrix.html
__global__ void ShowRGBfromXYZ(float4* target, float4* source, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= w || y >= h)	return;
	target[y*w + x] = XYZ2RGB(source[y*w + x]);
}

__global__ void ShowXYZfromRGB(float4* target, float4* source, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= w || y >= h)	return;
	target[y*w + x] = RGB2XYZ(source[y*w + x]);//out;
}

extern "C"
void GetModulationLayer(float4* target, int width, int height, int k)
{
	checkCudaErrors(hipMemcpy(target, dM, sizeof(float4)*width*height, hipMemcpyDeviceToHost));
}

extern "C"
void GetPrimaryLayer(float4* target, int width, int height, int ratio, int k)
{
	checkCudaErrors(hipMemcpy(target, dP, sizeof(float4)*(width/ratio)*(height/ratio), hipMemcpyDeviceToHost));
}

extern "C"
void GetSimulatedBLLayer(float4* target, int width, int height, int k)
{
	checkCudaErrors(hipMemcpy(target, dBL, sizeof(float4)*width*height, hipMemcpyDeviceToHost));
}

extern "C"
void GetRecLayer(float4* target, int width, int height, int k)
{
	dim3 GridSizeM((width + 32 - 1) / 32, (height + 32 - 1) / 32);
	dim3 BlockSize(32, 32);
	ShowRGBfromXYZ <<<GridSizeM, BlockSize >>>		(dTemp, dRec, width, height);
	checkCudaErrors(hipMemcpy(target, dTemp, sizeof(float4)*width*height, hipMemcpyDeviceToHost));
}